#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <set>
#define LLI long long int
// using namespace thrust;
using namespace std;

//*******************************************

// Write down the kernels here
__global__ void myMemSetInt(int *arr,int val,int size)
{
    if(threadIdx.x<size)
    arr[threadIdx.x]=val;
}




__global__ void dtanks(int *gxcoord, int * gycoord, int *gscore, int *ghealth, int *ghealthRead, int T, int N,
                         int *distance,  int *tanksLeft, int k){
    int cid=threadIdx.x;
    int sid=blockIdx.x;
    int did=(sid+k)%T;
    long long int x1=gxcoord[sid]; //source tanks
    long long int y1=gycoord[sid];
    long long int x2=gxcoord[did]; //destination tank
    long long int y2=gycoord[did];
    int dir=1;
    int dis=-2;
    

    if(did!=sid)
    {
        if(y2==y1)
        {
            if(x2<=x1)
            {
                dir=-1;
            }
        }
        else if(y1>y2)
        {
            dir=-1;
        }
    

        if(ghealthRead[sid]>0 && ghealthRead[cid]>0)
        {
            
                if(cid!=sid)
                {
                    long long int x=gxcoord[cid]; //current tank x coord
                    long long int y=gycoord[cid]; //current tank y coord
                    long long int lhs=((y-y1))*((x2-x1));
                    long long int rhs=((y2-y1))*((x-x1));

                    if(lhs==rhs) //condition to check whether the current tank lies in the direction of the fireline
                    {
                        if((dir>0 && (y1<y || (y1==y && x1<x))) || (dir<0 && (y1>y || (y==y1 && x1>x))))
                        {
                            dis=(abs(y-y1))+(abs(x-x1));
                            atomicMin(&distance[sid],dis);
                        }
                    }
                }
        }
    }
    __syncthreads();
    if(distance[sid]==dis)
    {
        atomicAdd(&gscore[sid],1);
        atomicSub(&ghealth[cid],1);
    }
}



__global__ void checktanks(int *ghealth, int *tanksleft)
{
    if(ghealth[threadIdx.x]<=0)
    {
        atomicSub(&tanksleft[0],1);
    }

    
}

// __global__ void SimulatingRounds(int *gpuk){
    
//     atomicAdd(&gpuk[0],1);
// }


//***********************************************


int main(int argc,char **argv)
{
    // printf("Hello Omlete");
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }
		

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    int *gxcoord, *gycoord, *gscore, *ghealth, *distance, *ghealthRead;
    
    // cudaDeviceSynchronize();

    hipMalloc(&gxcoord,sizeof(int)*T);
    hipMalloc(&gycoord,sizeof(int)*T);
    hipMalloc(&gscore,sizeof(int)*T);
    hipMalloc(&distance, sizeof(int)*T);
    // cudaMalloc(&CorrespondingIds, sizeof(int)*T);
    hipMalloc(&ghealth, sizeof(int)*T);
    hipMalloc(&ghealthRead, sizeof(int)*T);
    hipMemcpy(gxcoord,xcoord,sizeof(int)*T,hipMemcpyHostToDevice);
    hipMemcpy(gycoord,ycoord,sizeof(int)*T,hipMemcpyHostToDevice);
    myMemSetInt<<<1,1024>>>(gscore,0,T);
    myMemSetInt<<<1,1024>>>(ghealth,H,T);
    myMemSetInt<<<1,1024>>>(ghealthRead,H,T);
    myMemSetInt<<<1,1024>>>(distance,INT_MAX,T);
    
    


    //printing the health of tanks before war
    // int * host_health = (int *)malloc(sizeof(int)*T);
    // cudaMemcpy(host_health, ghealth, sizeof(int)*T, cudaMemcpyDeviceToHost);
    // printf("\n");
    // for(int i=0; i<T; i++){
    //     printf("%d ",host_health[i]);
    // }
    // printf("\n");

    int *tanksLeft;
    hipMalloc(&tanksLeft,sizeof(int));
    myMemSetInt<<<1,1024>>>(tanksLeft,T,1);
    int CTanks[1]={T};
    int k=0;

    while(CTanks[0]>1)
    {
        k++;
        // if(k%T!=0) //Allowed by Rupesh Nasre Sir
        // {
            
            dtanks<<<T,T>>>(gxcoord, gycoord, gscore, ghealth, ghealthRead, T, N, distance, tanksLeft, k);
            // cudaDeviceSynchronize();
            
            
            // printf("\n");
            // printf("Round %d:\n", k);
            // cudaMemcpy(score,gscore,sizeof(int)*T,cudaMemcpyDeviceToHost);
            // printf("Score: ");
            // for(int i=0; i<T; i++) printf("%d ",score[i]);
            // printf("\n");
            // printf("Health: ");
            // cudaMemcpy(host_health,ghealth,sizeof(int)*T,cudaMemcpyDeviceToHost);
            // for(int i=0; i<T; i++) printf("%d ", host_health[i]);


            myMemSetInt<<<1,1024>>>(distance,INT_MAX,T);
            // cudaDeviceSynchronize();
            checktanks<<<1,T>>>(ghealth,tanksLeft);
            hipMemcpy(&CTanks[0],tanksLeft,sizeof(int)*1,hipMemcpyDeviceToHost);
            myMemSetInt<<<1,1024>>>(tanksLeft,T,1);
            hipMemcpy(ghealthRead,ghealth,sizeof(int)*T,hipMemcpyDeviceToDevice);
            // printf("%d ",CTanks[0]);
            hipDeviceSynchronize();
        // }
    }
    
    hipMemcpy(score,gscore,sizeof(int)*T,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    // printf("\n");
    // for(int i=0; i<T; i++){
    //     printf("%d\n",score[i]);
    // }
    // printf("\n");
    //printing health after war
    // for(int i=0; i<T; i++){
    //     printf("%d ",host_health[i]);
    // }

    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}